
#include <hip/hip_runtime.h>
#include <iostream>
#include <bits/stdc++.h>
// #define n 13
// #define m 23
using namespace std;
#include <vector>
#include <algorithm>
#include <queue>

using namespace std;
__global__ void bfs_round_kernel(
    const int*  vertex,
    const int*  edges,
    const int*  frontier,
    int          k,
    int          n,
    int*         visited,
    int*         level,
    int*         next_frontier,
    int*         nextCount,
    int          depth
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= k) return;
    int u = frontier[tid];
    for (int e = vertex[u]; e < vertex[u+1]; e++) {
        int v = edges[e];
        if (atomicExch(&visited[v], 1) == 0) {
            level[v] = depth + 1;
            int pos = atomicAdd(nextCount, 1);
            if (pos < n) next_frontier[pos] = v;
        }
    }
}

// CPU-side “tail” of BFS
void cpu_bfs_round(
    const int*  frontier,
    int         frontier_size,
    int         start_idx,
    const int*  vertex,
    const int*  edges,
    int*        visited,
    int*        level,
    int*        next_frontier,
    int&        nextCount,
    int         depth
) {
    for (int idx = start_idx; idx < frontier_size; idx++) {
        int u = frontier[idx];
        for (int e = vertex[u]; e < vertex[u+1]; e++) {
            int v = edges[e];
            if (!visited[v]) {
                visited[v]    = 1;
                level[v]      = depth + 1;
                next_frontier[nextCount++] = v;
            }
        }
    }
}
void createNewCSR(
    int                       n,
    const int*                vertex,
    const int*                edges,
    const std::vector<int>&   W,
    const std::vector<int>&   S,
    int*                      new_vertex,
    int*&                     new_edges,
    int&                      new_m
) {
    int k = (int)W.size();

    // 1) Build a lookup to tell which vertices are sinks
    std::vector<bool> isSink(n, false);
    for (int u : S) {
        isSink[u] = true;
    }

    // 2) Build inverse‐map for W → new‐IDs [0..k-1]
    //    (we leave invPerm[v] = -1 for sinks)
    std::vector<int> invPerm(n, -1);
    for (int i = 0; i < k; i++) {
        invPerm[W[i]] = i;
    }

    // 3) First pass: count, per W‐row, how many *non‑sink* edges it has
    new_vertex[0] = 0;
    for (int i = 0; i < k; i++) {
        int oldV = W[i];
        int start = vertex[oldV];
        int end   = vertex[oldV+1];
        int cnt   = 0;
        for (int e = start; e < end; e++) {
            int nbr = edges[e];
            if (!isSink[nbr]) {
                ++cnt;
            }
        }
        new_vertex[i+1] = new_vertex[i] + cnt;
    }

    // 4) Allocate edges array
    new_m      = new_vertex[k];
    new_edges  = new int[new_m];

    // 5) Second pass: actually fill them in
    //    track per‐row fill position
    std::vector<int> fillPos(k, 0);
    for (int i = 0; i < k; i++) {
        int oldV    = W[i];
        int start   = vertex[oldV];
        int end     = vertex[oldV+1];
        int baseOff = new_vertex[i];

        for (int e = start; e < end; e++) {
            int nbr = edges[e];
            if (!isSink[nbr]) {
                // remap neighbor to its new‐ID
                int newNbr = invPerm[nbr];
                int pos    = baseOff + fillPos[i]++;
                new_edges[pos] = newNbr;
            }
        }
    }
}


void reorder(int n, int m, int* vertex, int* edges, vector<int>& W , vector<int>& S) {
     // Dynamic list
    int* N = new int[n]; // Dynamic array of size n
    int head = 0;
    int tail = 0;

    int* count = new int[n]();

    for (int i = 0; i < m; i++) {
        count[edges[i]]++;
    }

    int* V = new int[n];
    for (int i = 0; i < n; i++) {
        V[i] = count[i];
    }

    std::sort(V, V + n, std::greater<int>());

    /*for (int i = 0; i < n; i++) {
        cout << V[i] << endl;
    }
    cout << endl;*/
    W.resize(n, -1); // Resize W and initialize with -1
    vector<bool> inW(n, false); // To check if a vertex is in W
    vector<bool> inS(n, false); // To check if a vertex is in S

    // Sort vertices by in-degree in descending order
    vector<int> sortedVertices(n);
    iota(sortedVertices.begin(), sortedVertices.end(), 0);
    sort(sortedVertices.begin(), sortedVertices.end(), [&](int a, int b) {
        return count[a] > count[b];
    });

    for (int vi : sortedVertices) {
        if (!inW[vi] && !inS[vi]) {
            bool hasOutNeighbors = (vertex[vi + 1] > vertex[vi]);
            if (hasOutNeighbors) {
                W[tail++] = vi;
                inW[vi] = true;
            } else {
                S.push_back(vi);
                inS[vi] = true;
            }

            while (head < tail) {
                int vcur = W[head++];
                vector<int> neighbors;

                // Collect all out-neighbors of vcur
                for (int j = vertex[vcur]; j < vertex[vcur + 1]; j++) {
                    neighbors.push_back(edges[j]);
                }

                // Sort neighbors by in-degree in descending order
                sort(neighbors.begin(), neighbors.end(), [&](int a, int b) {
                    return count[a] > count[b];
                });

                for (int vj : neighbors) {
                    if (!inW[vj] && !inS[vj]) {
                        bool vjHasOutNeighbors = (vertex[vj + 1] > vertex[vj]);
                        if (vjHasOutNeighbors) {
                            W[tail++] = vj;
                            inW[vj] = true;
                        } else {
                            S.push_back(vj);
                            inS[vj] = true;
                        }
                    }
                }
            }
        }
    }

    // Resize W to the actual size
    W.resize(tail);

    // Print W at the end
    /*cout << "W: ";
    for (int i = 0; i < tail; i++) {
        cout << W[i] << " ";
    }
    cout << endl;*/

    delete[] N;
    delete[] count;
    delete[] V;
}


void createCSR(int N, int arr1[], int arr2[], int edgeCount, int* vertex, int* edges) {
    // Initialize vertex array with zeros
    for (int i = 0; i <= N; i++) {
        vertex[i] = 0;
    }

    // Count the number of edges for each vertex
    for (int i = 0; i < edgeCount; i++) {
        if (arr1[i] >= 0 && arr1[i] < N) {
            vertex[arr1[i] + 1]++;
        } else {
            std::cerr << "Error: arr1[" << i << "] is out of bounds." << std::endl;
            exit(1);
        }
    }

    // Compute prefix sum to determine starting indices in edges array
    for (int i = 1; i <= N; i++) {
        vertex[i] += vertex[i - 1];
    }

    // Fill the edges array
    for (int i = 0; i < edgeCount; i++) {
        int u = arr1[i];
        int index = vertex[u]++;
        edges[index] = arr2[i];
    }

    // Restore vertex array to correct starting indices
    for (int i = N; i > 0; i--) {
        vertex[i] = vertex[i - 1];
    }
    vertex[0] = 0;
}



int main(int argc, char* argv[]) {
    int n=0;
    int m=0;

    #include <fstream>
    #include <sstream>

    if (argc < 4) {
        std::cerr << "Usage: " << argv[0] << " <input_file> <k> <src>" << std::endl;
        return 1;
    }

    std::ifstream inputFile(argv[1]);
    if (!inputFile.is_open()) {
        std::cerr << "Error: Could not open file " << argv[1] << std::endl;
        return 1;
    }

    int k = std::stoi(argv[2]);
    if (k <= 0) {
        std::cerr << "Error: k must be a positive integer." << std::endl;
        return 1;
    }

    int src = std::stoi(argv[3]);
    if (src < 0) {
        std::cerr << "Error: src must be a non-negative integer." << std::endl;
        return 1;
    }

    inputFile >> n >> m;
    int* arr1 = new int[m];
    int* arr2 = new int[m];




    for (int i = 0; i < m; ++i) {
        int u, v, t, x;

        inputFile >> u >> v >> t >> x;
        //cout << u << "  " << v <<endl;
        arr1[i] = u;
        arr2[i] = v;
    }

    inputFile.close();

    int* vertex = new int[n + 1];
    int* edges = new int[m];

    createCSR(n, arr1, arr2, m, vertex, edges);


    vector<int> S;
    std::vector<int> W;
    reorder(n, m, vertex, edges, W,S);

    // Print W
    cout << "W: ";
    for (int i = 0; i < W.size(); i++) {
        cout << W[i] << " ";
    }
    cout << endl;

    // Print S
    cout <<endl << "S: " << endl;
    for (int i = 0; i < S.size(); i++) {
        cout << S[i] << " ";
    }
    cout << endl;


    int* new_vertex = new int[n+1];
    int* new_edges  = new int[m];

createNewCSR(n, vertex, edges, W, S, new_vertex, new_edges, m);




    // Print new_vertex
    cout << "new_vertex: ";
    for (int i = 0; i <= n; i++) {
        cout << new_vertex[i] << " ";
    }
    cout << endl;

    // Print new_edges
    cout << "new_edges: ";
    for (int i = 0; i < m; i++) {
        cout << new_edges[i] << " ";
    }
    cout << endl;

    int *visited_cpu = new int[n]();      // zero‐inited
    int *level_cpu   = new int[n];
    fill(level_cpu, level_cpu + n, -1);
    visited_cpu[src] = 1;
    level_cpu[src]   = 0;

    int *frontier_cpu      = new int[n];
    int  frontier_size     = 1;
    frontier_cpu[0]        = src;
    int *next_frontier_cpu = new int[n];
    int  nextCount_cpu     = 0;

    // --- Allocate+copy to GPU ---
    int *d_vertex, *d_edges;
    int *d_frontier, *d_visited, *d_level, *d_next_frontier, *d_nextCount;
    hipMalloc(&d_vertex,      (n+1)*sizeof(int));
    hipMalloc(&d_edges,       m*sizeof(int));
    hipMalloc(&d_frontier,    n*sizeof(int));
    hipMalloc(&d_visited,     n*sizeof(int));
    hipMalloc(&d_level,       n*sizeof(int));
    hipMalloc(&d_next_frontier, n*sizeof(int));
    hipMalloc(&d_nextCount,   sizeof(int));

    hipMemcpy(d_vertex, new_vertex, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edges,  new_edges,   m*sizeof(int),   hipMemcpyHostToDevice);

    // zero‐init device visited/level
    hipMemset(d_visited, 0, n*sizeof(int));
    hipMemset(d_level,  -1, n*sizeof(int));

    // mark source on device
    int one = 1;
    hipMemcpy(&d_frontier[0], &src,         sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&d_visited[src], &one,        sizeof(int), hipMemcpyHostToDevice);
    int zero = 0;
    hipMemcpy(&d_level[src],   &zero,       sizeof(int), hipMemcpyHostToDevice);

    int depth = 0;
    const int THREADS = 256;

    // --- BFS main loop ---
    while (frontier_size > 0) {
        // reset GPU’s nextCount = 0
        hipMemcpy(d_nextCount, &zero, sizeof(int), hipMemcpyHostToDevice);

        // copy current frontier to device
        hipMemcpy(d_frontier, frontier_cpu,
                   frontier_size*sizeof(int), hipMemcpyHostToDevice);

        // GPU handles up to k_gpu = min(k_user, frontier_size)
        int k_gpu = min(k, frontier_size);
        int blocks = (k_gpu + THREADS - 1)/THREADS;
        bfs_round_kernel<<<blocks, THREADS>>>(
            d_vertex, d_edges, d_frontier, k_gpu, n,
            d_visited, d_level, d_next_frontier, d_nextCount, depth
        );
        hipDeviceSynchronize();

        // fetch how many the GPU actually enqueued
        hipMemcpy(&k_gpu, d_nextCount, sizeof(int), hipMemcpyDeviceToHost);

        // fetch those frontier items back to host
        hipMemcpy(next_frontier_cpu, d_next_frontier,
                   k_gpu * sizeof(int), hipMemcpyDeviceToHost);

        // CPU handles the tail [k_gpu .. frontier_size)
        nextCount_cpu = 0;
        if (frontier_size > k_gpu) {
            cpu_bfs_round(
                frontier_cpu, frontier_size, k_gpu,
                new_vertex, new_edges,
                visited_cpu, level_cpu,
                next_frontier_cpu, nextCount_cpu,
                depth
            );
        }

        // total new frontier size = GPU’s + CPU’s
        int new_frontier_size = k_gpu + nextCount_cpu;

        // swap frontiers: next_frontier_cpu is now our frontier_cpu
        memcpy(frontier_cpu, next_frontier_cpu,
               new_frontier_size * sizeof(int));
        frontier_size = new_frontier_size;

        depth++;
    }

    // --- copy final levels back from GPU and print both sides ---
    int* final_level_gpu = new int[n];
    hipMemcpy(final_level_gpu, d_level, n*sizeof(int), hipMemcpyDeviceToHost);

    cout << "Depth by GPU/CPU hybrid:\n";
    for (int i = 0; i < n; i++)
        cout << final_level_gpu[i] << " ";
    cout << endl;

    // cleanup…
    hipFree(d_vertex); hipFree(d_edges);
    hipFree(d_frontier); hipFree(d_visited);
    hipFree(d_level);   hipFree(d_next_frontier);
    hipFree(d_nextCount);

    delete[] new_vertex; delete[] new_edges;
    delete[] visited_cpu; delete[] level_cpu;
    delete[] frontier_cpu; delete[] next_frontier_cpu;
    delete[] final_level_gpu;

    return 0;

    return 0;
}
